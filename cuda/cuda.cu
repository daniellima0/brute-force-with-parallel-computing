#include "hip/hip_runtime.h"
// CUDA

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <unistd.h>

// 97 to 122 use only lowercase letters
// 65 to 90 use only capital letters
// 48 to 57 use only numbers
#define START_CHAR 48
#define END_CHAR 122
#define MAXIMUM_PASSWORD 20

__global__ void bruteForce(char *pass, unsigned int size);
__host__ __device__ long long my_pow(long long x, int y);
__host__ __device__ unsigned int my_strlen(char *s);

int main(int argc, char **argv) {
    size_t size = sizeof(char) * MAXIMUM_PASSWORD;
    char *password;
    hipMallocManaged(&password, size);
    strcpy(password, argv[1]);

    size_t threadsPerBlock = 1024;
    size_t numberOfBlocks = 40;

    printf("Estamos tentando quebrar: %s\n", password);

    unsigned int numberOfCharacters = my_strlen(password);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    bruteForce<<<numberOfBlocks, threadsPerBlock>>>(password, numberOfCharacters);
    hipEventRecord(stop);

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    printf("\n%1.2f seconds\n", milliseconds / 1000);

    return 0;
}

/*Check and increase the digits if you don't find the password...*/
__global__ void bruteForce(char *pass, unsigned int size) {
    int pass_b26[MAXIMUM_PASSWORD];
    long long int j;
    long long int pass_decimal = 0;
    int base = END_CHAR - START_CHAR + 2;

    for (int i = 0; i < size; i++) {
        pass_b26[i] = (int)pass[i] - START_CHAR + 1;  //+1 pois o vazio é o zero e o 'a' é 1
    }

    for (int i = size - 1; i > -1; i--) {
        pass_decimal += (long long int)pass_b26[i] * my_pow(base, i);
    }

    long long int max = my_pow(base, 9);
    char s[10];
    for (j = blockIdx.x * blockDim.x + threadIdx.x; j < max; j += blockDim.x * gridDim.x) {
        if (j == pass_decimal) {
            printf("Encontrou o password!\n");
            int index = 0;

            printf("O número que estamos tentando encontrar (password na base decimal): %lli\n", j);
            while (j > 0) {
                s[index++] = START_CHAR + j % base - 1;
                j /= base;
            }
            s[index] = '\0';
            printf("Password encontrado: %s\n", s);
            break;
        } else if (j > pass_decimal) {
            break;
        }
    }
}

__host__ __device__ long long my_pow(long long x, int y) {
    long long res = 1;
    if (y == 0)
        return res;
    else
        return x * my_pow(x, y - 1);
}

unsigned int my_strlen(char *palavra) {
    int i = 0;

    while (palavra[i] != '\0') {
        i++;
    }

    return i;
}